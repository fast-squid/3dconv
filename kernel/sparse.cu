#include "hip/hip_runtime.h"
#include "mat.h"
#include <iostream>
#include <stdlib.h>

#define ERROR_CHECK \
{\
	hipError_t err = hipGetLastError(); \
	if ( hipSuccess != err ) \
	{\
		printf("[%s:%d]CUDA ERROR : %s\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
	}\
}
__device__ __constant__ float eps = 0.0001;

// implementation without convolution parameter(padding, stride, groups, dilation)
__global__ void cube_to_coo(int input_d, int input_h, int input_w, float* input_data,
		int output_d, int output_h, int output_w, COO* coo, int *nnz, int stride)
{
    // block[8] : 0 ~ 7
	//printf(".\n");	
	if(threadIdx.x <5 && threadIdx.y < 5 && threadIdx.z <5)
	{
		
		int input_w_idx = (blockIdx.x % output_w)*stride + threadIdx.x;
		int input_h_idx = ((blockIdx.x / output_w) % output_h)*stride + threadIdx.y;
		int input_d_idx = (((blockIdx.x / output_w) / output_h) % output_d)*stride + threadIdx.z;

		int input_idx =  input_d_idx*(input_h*input_w)
			+ input_h_idx*(input_w)
			+ input_w_idx;
		if(input_data[input_idx]>eps)
		{
			int global_idx = atomicAdd(nnz, 1);
			coo[global_idx].row = threadIdx.z*(25)
				+ threadIdx.y*5
				+ threadIdx.x;
			coo[global_idx].col = blockIdx.x;
			coo[global_idx].val = input_data[input_idx];
		}
	}
	
}

int compare(const void* a, const void* b)
{
	COO x = *(COO*)a;
	COO y = *(COO*)b;
	if(x.col == y.col) return x.row - y.row;
	else return x.col - y.col;
}

int compare2(const void* a, const void* b)
{
	COO x = *(COO*)a;
	COO y = *(COO*)b;
	if(x.row == y.row) return x.col - y.col;
	else return x.row - y.row;
}
void cube_to_coo_cuda(Mat& input, Mat& filter, Param& p)
{
	int output_N = 1;
	int output_C = filter.N;
	int output_D = 1+(input.D - filter.D + 2*p.padding)/p.stride;
	int output_H = 1+(input.H - filter.H + 2*p.padding)/p.stride;
	int output_W = 1+(input.W - filter.W + 2*p.padding)/p.stride;
	
	input.row_num = filter.C*filter.D*filter.H*filter.W;
	input.col_num = output_D*output_H*output_W;

	printf("input shape  : (%d,%d,%d,%d,%d)\n",input.N, input.C, input.D, input.H, input.W);
	printf("output shape : (%d,%d,%d,%d,%d)\n",output_N, output_C, output_D, output_H, output_W);
	printf("im2col shape : (%d,%d)\n",input.row_num, input.col_num);
	// coo format
	int* nnz_d;

	hipMalloc((void**)&input.coo_dev,sizeof(COO)*400000);
	hipMalloc((void**)&nnz_d, sizeof(int));
	hipMemset(nnz_d, 0, sizeof(int));
	
	// 3D-input
	int input_size = input.N*input.C*input.D*input.H*input.W;
	hipMalloc((void**)&input.data_dev, sizeof(float)*input_size);
	hipMemcpy(input.data_dev, input.data, sizeof(float)*input_size,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	int block_num = output_D*output_H*output_W;
	dim3 block_size(8,8,8);
	
	cube_to_coo<<<block_num, block_size>>>
		(input.D, input.H, input.W, input.data_dev,
		 output_D, output_H, output_W,input.coo_dev,
		 nnz_d,p.stride);
	ERROR_CHECK;	

	hipMemcpy(&input.nnz,nnz_d, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	input.coo = new COO[input.nnz];
	hipMemcpy(input.coo, input.coo_dev, input.nnz*sizeof(COO),  hipMemcpyDeviceToHost);
	qsort(input.coo, input.nnz, sizeof(COO),compare);
	hipMemcpy(input.coo_dev, input.coo, input.nnz*sizeof(COO),  hipMemcpyHostToDevice);

	// cudafree
	hipFree(input.data_dev);
}

__global__ void coo_2_csr(COO* coo, 
		int* ptr, int *idx, float* val,
		int nnz)
{
    int global_tid = threadIdx.x+blockDim.x*blockIdx.x;
	if(global_tid<nnz)
	{
		idx[global_tid] = coo[global_tid].col;
		val[global_tid] = coo[global_tid].val;
		atomicAdd(&ptr[coo[global_tid].row+1], 1);
	}
}

void coo_to_csr_cuda(Mat& input)
{
	hipMalloc((void**)&input.ptr_dev, sizeof(int)*(input.row_num+1));
	hipMalloc((void**)&input.idx_dev, sizeof(int)*input.nnz);
	hipMalloc((void**)&input.val_dev, sizeof(float)*input.nnz);
	hipMemset(input.ptr_dev, 0, sizeof(int)*(input.row_num+1));

	int block_num = input.nnz/1024+1;
	int block_size = 1024;
	
	coo_2_csr<<<block_num, block_size>>>(input.coo_dev,
			input.ptr_dev, input.idx_dev, input.val_dev,
			input.nnz);
	ERROR_CHECK;	
	
	input.ptr = new int[input.row_num+1];
	input.idx = new int[input.nnz];
	input.val = new float[input.nnz];

	hipMemcpy(input.ptr, input.ptr_dev, sizeof(int)*(input.row_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(input.idx, input.idx_dev, sizeof(int)*input.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(input.val, input.val_dev, sizeof(float)*input.nnz,hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	for(int i=0;i<input.row_num;i++)
	{
		input.ptr[i+1] += input.ptr[i];
	}
	printf("nnz %d %d\n",input.nnz, input.ptr[input.row_num]);
	hipMemcpy(input.ptr_dev, input.ptr, sizeof(int)*(input.row_num+1),hipMemcpyHostToDevice);
}

__global__ void coo_2_csc(COO* coo, 
		int* ptr, int *idx, float* val,
		int nnz)
{
    int global_tid = threadIdx.x+blockDim.x*blockIdx.x;
	if(global_tid<nnz)
	{
		idx[global_tid] = coo[global_tid].row;
		val[global_tid] = coo[global_tid].val;
		atomicAdd(&ptr[coo[global_tid].col+1], 1);
	}
}

void coo_to_csc_cuda(Mat& input, int& number_of_non_zero_vectors, int *non_zero_vectors)
{
	hipMalloc((void**)&input.ptr_dev, sizeof(int)*(input.col_num+1));
	hipMalloc((void**)&input.idx_dev, sizeof(int)*input.nnz);
	hipMalloc((void**)&input.val_dev, sizeof(float)*input.nnz);
	hipMemset(input.ptr_dev, 0, sizeof(int)*(input.col_num+1));

	int block_num = input.nnz/1024+1;
	int block_size = 1024;
	
	coo_2_csc<<<block_num, block_size>>>(input.coo_dev,
			input.ptr_dev, input.idx_dev, input.val_dev,
			input.nnz);
	ERROR_CHECK;	
	
	input.ptr = new int[input.col_num+1];
	input.idx = new int[input.nnz];
	input.val = new float[input.nnz];

	hipMemcpy(input.ptr, input.ptr_dev, sizeof(int)*(input.col_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(input.idx, input.idx_dev, sizeof(int)*input.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(input.val, input.val_dev, sizeof(float)*input.nnz,hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	for(int i=0;i<input.col_num;i++)
	{
		input.ptr[i+1] += input.ptr[i];
		if(input.ptr[i+1]-input.ptr[i])
		{
			non_zero_vectors[number_of_non_zero_vectors] = i;
			number_of_non_zero_vectors++;
		}
	}

	printf("nnz %d %d\n",input.nnz, input.ptr[input.col_num]);
	hipMemcpy(input.ptr_dev, input.ptr, sizeof(int)*(input.col_num+1),hipMemcpyHostToDevice);
}

/*
 * Function: dense_sparse_mm
 * -----------------------------------------------------------------------
 * computes matrix multiplication  C = AxB corresponding to convolution 
 * Parameters ------------------------------------------------------------
 * matrix A (dense matrix A correspoinding to filter)
 * 	a_height : height of matrix A
 *	a_width  : width of matrix A 
 * 	a_val    : value array of matrix A
 * matrix B (CSC formatted B corresponding to input feature map) 
 *	b_ptr	 : row pointer array of matrix B 
 *	b_idx	 : column index array of matrix B
 *	b_val	 : value array of matrix B
 * non_zero_vectors : non-zero column vector idx array of matrix B
 * returns ---------------------------------------------------------------
 * matrix C (CSC forammted C correspoint to output feature map)
 * 	 c_ptr, c_idx, c_val : same as B
 */
__global__ void dense_sparse_mm(int a_height, int a_width, float* a_val,
		int* b_ptr, int* b_idx, float* b_val,
		int* c_ptr, int* c_idx, float* c_val,
		int* non_zero_vectors)
{
	int b_col_idx = non_zero_vectors[blockIdx.x];
	int row_offset = b_ptr[b_col_idx];
	int nnz = b_ptr[b_col_idx + 1] - b_ptr[b_col_idx];
	
	__shared__ float smem_a_row[4][32];
	__shared__ int smem_b_row[128];
	__shared__ float smem_b_val[128];

	// load a's column
	const int tile_width = 4;
	const int tile_height = 32;
	int total_tile = nnz/tile_width; // tile size : (32 * 4)
	int remainings = nnz%tile_width;
	
	for(int tid = threadIdx.x; tid < nnz; tid+=blockDim.x)
	{
		smem_b_row[tid] = b_idx[row_offset + tid];
		smem_b_val[tid] = b_val[row_offset + tid];
	}
	__syncthreads();

	// compute tiled matrix multiplication
	float val = 0;
	int tile_id = 0;
	for(tile_id = 0; tile_id < total_tile; tile_id++)
	{	
		int tile_offset = tile_width * tile_id;
		smem_a_row[threadIdx.x][0] 
			= a_val[a_width*threadIdx.x + smem_b_row[tile_offset+0]];
		smem_a_row[threadIdx.x][1] 
			= a_val[a_width*threadIdx.x + smem_b_row[tile_offset+1]];
		smem_a_row[threadIdx.x][2] 
			= a_val[a_width*threadIdx.x + smem_b_row[tile_offset+2]];
		smem_a_row[threadIdx.x][3] 
			= a_val[a_width*threadIdx.x + smem_b_row[tile_offset+3]];
		__syncthreads();
		
		for(int a_ridx = threadIdx.x ; a_ridx < a_height; a_ridx += blockDim.x)
		{
			val += smem_a_val[a_ridx][0] * smem_b_val[tile_offset + 0];
			val += smem_a_val[a_ridx][1] * smem_b_val[tile_offset + 1];
			val += smem_a_val[a_ridx][2] * smem_b_val[tile_offset + 2];
			val += smem_a_val[a_ridx][3] * smem_b_val[tile_offset + 3];
		}
	}
	
	// compute remaining matrix multiplication
	for(int r = 0; r < remainings; r++)
	{
		int tile_offset = tile_id*tile_width;
		smem_a_row[threadIdx.x][r] 
			= a_val[a_width*threadIdx.x + smem_b_row[tile_offset + r]];
	}
	for(int r = 0; r < remainings; r++)
	{
		val += smem_a_val[threadIdx.x][r] * smem_b_val[tile_offset + r];
	}
	c_idx[height*blockIdx.x + threadIdx.x] 
	c_val[height*blockIdx.x + threadIdx.x] = val;
	c_ptr[col_idx+1] = 32;
}

void call_when_model_loaded(Mat& filter)
{
	int size = filter.N*filter.C*filter.D*filter.H*filter.W;
	hipMalloc((void**)&filter.data_dev, sizeof(int)*size);
	hipMemcpy(filter.data_dev, filter.data, sizeof(int)*size, hipMemcpyHostToDevice);
	printf("filter size %d\n",size);
}

void dense_sparse_mm_cuda(Mat& input, Mat& filter, Mat& output,
		int number_of_non_zero_vectors, int* non_zero_vectors)
{
	int block_num = number_of_non_zero_vectors;
	int block_size = 32;
	
	int a_height = filter.N;
	int a_width = filter.C*filter.D*filter.H*filter.W;
	int b_height = input.row_num;
	int b_width = input.col_num;

	int* non_zero_vectors_dev;
	hipMalloc((void**)&non_zero_vectors_dev, sizeof(int)*number_of_non_zero_vectors);
	hipMemcpy(non_zero_vectors_dev, non_zero_vectors, sizeof(int)*number_of_non_zero_vectors, hipMemcpyHostToDevice);
	call_when_model_loaded(filter);

	output.row_num = a_height;
	output.col_num = b_width;
	output.nnz = a_height*number_of_non_zero_vectors;

	hipMalloc((void**)&output.ptr_dev, sizeof(int)*(output.col_num+1));
	hipMalloc((void**)&output.idx_dev, sizeof(int)*output.nnz);
	hipMalloc((void**)&output.val_dev, sizeof(float)*output.nnz);
	hipMemset(output.ptr_dev, 0, sizeof(int)*(output.col_num+1));
	printf("%d %d\n",a_height, a_width);
	printf("%d %d\n",b_height, b_width);
	printf("%d %d %d\n",output.row_num, output.col_num, output.nnz);
	printf("bs %d bn %d\n",block_size, block_num);	
	
	dense_sparse_mm<<<block_num, block_size >>>(a_height, a_width,filter.data_dev,
		input.ptr_dev, input.idx_dev, input.val_dev,
		output.ptr_dev,  output.idx_dev, output.val_dev,
		non_zero_vectors_dev);
	
	output.ptr = new int[output.col_num+1];
	output.idx = new int[output.nnz];
	output.val = new float[output.nnz];
	
	hipMemcpy(output.ptr, output.ptr_dev, sizeof(int)*(output.col_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(output.idx, output.idx_dev, sizeof(int)*output.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(output.val, output.val_dev, sizeof(float)*output.nnz,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	Mat temp;
	temp.row_num = output.row_num;
	temp.col_num = output.col_num;
	temp.coo = new COO[output.nnz];
	int temp_idx = 0;
	for(int i=0; i<output.col_num; i++)
	{
		output.ptr[i+1] += output.ptr[i];
		int nnz_per_col = output.ptr[i+1] - output.ptr[i] ;
		int offset = output.ptr[i];
		for(int j=0;j<nnz_per_col;j++)
		{
			temp.coo[temp_idx].row = output.idx[offset+j];
			temp.coo[temp_idx].col = i;
			temp.coo[temp_idx].val = output.val[offset+j];
			temp_idx++;
		}
	}
	temp.nnz = output.ptr[output.col_num];
	qsort(temp.coo, temp.nnz, sizeof(COO),compare2);
	print_coo(temp);
	
}

void sparse_conv_cuda(Mat& input, Mat& filter, Param& p, Mat& output)
{
	cube_to_coo_cuda(input, filter, p);
	int* non_zero_vectors = new int[input.col_num+1];
	int number_of_non_zero_vectors = 0;
	coo_to_csc_cuda(input, number_of_non_zero_vectors, non_zero_vectors);
	printf("number of non zero vectors : %d\n",number_of_non_zero_vectors);
	dense_sparse_mm_cuda(input, filter, output, number_of_non_zero_vectors, non_zero_vectors);

	
}
