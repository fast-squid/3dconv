#include "hip/hip_runtime.h"
#include "mat.h"
#include <iostream>
#include <stdlib.h>

#define ERROR_CHECK \
{\
	hipError_t err = hipGetLastError(); \
	if ( hipSuccess != err ) \
	{\
		printf("[%s:%d]CUDA ERROR : %s\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
	}\
}
__device__ __constant__ float eps = 0.0001;

// implementation without convolution parameter(padding, stride, groups, dilation)
__global__ void cube_to_coo(int input_d, int input_h, int input_w, float* input_data,
		int output_d, int output_h, int output_w, COO* coo, int *nnz, int stride)
{
    // block[8] : 0 ~ 7
	//printf(".\n");	
	if(threadIdx.x <5 && threadIdx.y < 5 && threadIdx.z <5)
	{
		
		int input_w_idx = (blockIdx.x % output_w)*stride + threadIdx.x;
		int input_h_idx = ((blockIdx.x / output_w) % output_h)*stride + threadIdx.y;
		int input_d_idx = (((blockIdx.x / output_w) / output_d) % output_d)*stride + threadIdx.z;

		int input_idx =  input_d_idx*(input_h*input_w)
			+ input_h_idx*(input_w)
			+ input_w_idx;
		if(input_data[input_idx]>eps)
		{
			int global_idx = atomicAdd(nnz, 1);
			coo[global_idx].row = threadIdx.z*(25)
				+ threadIdx.y*5
				+ threadIdx.x;
			coo[global_idx].col = blockIdx.x;
			coo[global_idx].val = input_data[input_idx];
		}
	}
	
}

int compare(const void* a, const void* b)
{
	COO x = *(COO*)a;
	COO y = *(COO*)b;
	if(x.col == y.col) return x.row - y.row;
	else return x.col - y.col;
}

int compare2(const void* a, const void* b)
{
	COO x = *(COO*)a;
	COO y = *(COO*)b;
	if(x.row == y.row) return x.col - y.col;
	else return x.row - y.row;
}
void cube_to_coo_cuda(Mat& input, Mat& filter, Param& p)
{
	int output_N = 1;
	int output_C = filter.N;
	int output_D = 1+(input.D - filter.D + 2*p.padding)/p.stride;
	int output_H = 1+(input.H - filter.H + 2*p.padding)/p.stride;
	int output_W = 1+(input.W - filter.W + 2*p.padding)/p.stride;
	
	input.row_num = filter.C*filter.D*filter.H*filter.W;
	input.col_num = output_D*output_H*output_W;

	printf("input shape  : (%d,%d,%d,%d,%d)\n",input.N, input.C, input.D, input.H, input.W);
	printf("output shape : (%d,%d,%d,%d,%d)\n",output_N, output_C, output_D, output_H, output_W);
	printf("im2col shape : (%d,%d)\n",input.row_num, input.col_num);
	// coo format
	int* nnz_d;

	hipMalloc((void**)&input.coo_dev,sizeof(COO)*400000);
	hipMalloc((void**)&nnz_d, sizeof(int));
	hipMemset(nnz_d, 0, sizeof(int));
	
	// 3D-input
	int input_size = input.N*input.C*input.D*input.H*input.W;
	hipMalloc((void**)&input.data_dev, sizeof(float)*input_size);
	hipMemcpy(input.data_dev, input.data, sizeof(float)*input_size,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	int block_num = output_D*output_H*output_W;
	dim3 block_size(8,8,8);
	
	cube_to_coo<<<block_num, block_size>>>
		(input.D, input.H, input.W, input.data_dev,
		 output_D, output_H, output_W,input.coo_dev,
		 nnz_d,p.stride);
	ERROR_CHECK;	

	hipMemcpy(&input.nnz,nnz_d, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	input.coo = new COO[input.nnz];
	hipMemcpy(input.coo, input.coo_dev, input.nnz*sizeof(COO),  hipMemcpyDeviceToHost);
	qsort(input.coo, input.nnz, sizeof(COO),compare);
	hipMemcpy(input.coo_dev, input.coo, input.nnz*sizeof(COO),  hipMemcpyHostToDevice);

	// cudafree
	hipFree(input.data_dev);
}

__global__ void coo_2_csr(COO* coo, 
		int* ptr, int *idx, float* val,
		int nnz)
{
    int global_tid = threadIdx.x+blockDim.x*blockIdx.x;
	if(global_tid<nnz)
	{
		idx[global_tid] = coo[global_tid].col;
		val[global_tid] = coo[global_tid].val;
		atomicAdd(&ptr[coo[global_tid].row+1], 1);
	}
}

void coo_to_csr_cuda(Mat& input)
{
	hipMalloc((void**)&input.ptr_dev, sizeof(int)*(input.row_num+1));
	hipMalloc((void**)&input.idx_dev, sizeof(int)*input.nnz);
	hipMalloc((void**)&input.val_dev, sizeof(float)*input.nnz);
	hipMemset(input.ptr_dev, 0, sizeof(int)*(input.row_num+1));

	int block_num = input.nnz/1024+1;
	int block_size = 1024;
	
	coo_2_csr<<<block_num, block_size>>>(input.coo_dev,
			input.ptr_dev, input.idx_dev, input.val_dev,
			input.nnz);
	ERROR_CHECK;	
	
	input.ptr = new int[input.row_num+1];
	input.idx = new int[input.nnz];
	input.val = new float[input.nnz];

	hipMemcpy(input.ptr, input.ptr_dev, sizeof(int)*(input.row_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(input.idx, input.idx_dev, sizeof(int)*input.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(input.val, input.val_dev, sizeof(float)*input.nnz,hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	for(int i=0;i<input.row_num;i++)
	{
		input.ptr[i+1] += input.ptr[i];
	}
	printf("nnz %d %d\n",input.nnz, input.ptr[input.row_num]);
	hipMemcpy(input.ptr_dev, input.ptr, sizeof(int)*(input.row_num+1),hipMemcpyHostToDevice);
}

__global__ void coo_2_csc(COO* coo, 
		int* ptr, int *idx, float* val,
		int nnz)
{
    int global_tid = threadIdx.x+blockDim.x*blockIdx.x;
	if(global_tid<nnz)
	{
		idx[global_tid] = coo[global_tid].row;
		val[global_tid] = coo[global_tid].val;
		atomicAdd(&ptr[coo[global_tid].col+1], 1);
	}
}

void coo_to_csc_cuda(Mat& input, int& number_of_non_zero_vectors, int *non_zero_vectors)
{
	hipMalloc((void**)&input.ptr_dev, sizeof(int)*(input.col_num+1));
	hipMalloc((void**)&input.idx_dev, sizeof(int)*input.nnz);
	hipMalloc((void**)&input.val_dev, sizeof(float)*input.nnz);
	hipMemset(input.ptr_dev, 0, sizeof(int)*(input.col_num+1));

	int block_num = input.nnz/1024+1;
	int block_size = 1024;
	
	coo_2_csc<<<block_num, block_size>>>(input.coo_dev,
			input.ptr_dev, input.idx_dev, input.val_dev,
			input.nnz);
	ERROR_CHECK;	
	
	input.ptr = new int[input.col_num+1];
	input.idx = new int[input.nnz];
	input.val = new float[input.nnz];

	hipMemcpy(input.ptr, input.ptr_dev, sizeof(int)*(input.col_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(input.idx, input.idx_dev, sizeof(int)*input.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(input.val, input.val_dev, sizeof(float)*input.nnz,hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	for(int i=0;i<input.col_num;i++)
	{
		input.ptr[i+1] += input.ptr[i];
		if(input.ptr[i+1]-input.ptr[i])
		{
			non_zero_vectors[number_of_non_zero_vectors] = i;
			number_of_non_zero_vectors++;
		}
	}

	printf("nnz %d %d\n",input.nnz, input.ptr[input.col_num]);
	hipMemcpy(input.ptr_dev, input.ptr, sizeof(int)*(input.col_num+1),hipMemcpyHostToDevice);
}



__global__ void dense_sparse_mm(int a_height, int a_width, float* a_val,
		int* b_ptr, int* b_idx, float* b_val,
		int* c_ptr, int* c_idx, float* c_val,
		int* non_zero_vectors)
{
	int col_idx = non_zero_vectors[blockIdx.x];
	int row_offset = b_ptr[col_idx];
	int nnz = b_ptr[col_idx + 1] - b_ptr[col_idx];
	
	// load B's row idx to shared memory
	__shared__ int smem_b_row[1024];
	__shared__ int smem_b_val[1024];

	for(int tid = threadIdx.x; tid < nnz; tid+=blockDim.x)
	{
		smem_b_row[tid] = b_idx[row_offset + tid];
		smem_b_val[tid] = b_val[row_offset + tid];
	}
	__syncthreads();

	__shared__ float smem_c_val[1024];


	for(int a_h = threadIdx.x; a_h < a_height; a_h+=blockDim.x)
	{
		int idx = smem_b_row[a_h];
		for(int idx = 0; idx < nnz; idx++)
		{
			smem_c_val[a_h] += a_val[a_h*a_width + idx] * smem_b_val[idx];
		}
	}
	c_ptr[col_idx+1] = a_height;
	for(int tid = threadIdx.x; tid < a_height; tid += blockDim.x)
	{
		c_val[a_height*blockIdx.x+tid] = smem_c_val[tid];
		c_idx[a_height*blockIdx.x+tid] = tid;
	}
	 
}

void call_when_model_loaded(Mat& filter)
{
	int size = filter.N*filter.C*filter.D*filter.H*filter.W;
	hipMalloc((void**)&filter.data_dev, sizeof(int)*size);
	hipMemcpy(filter.data_dev, filter.data, sizeof(int)*size, hipMemcpyHostToDevice);
	printf("filter size %d\n",size);
}

void dense_sparse_mm_cuda(Mat& input, Mat& filter, Mat& output,
		int number_of_non_zero_vectors, int* non_zero_vectors)
{
	int block_num = number_of_non_zero_vectors;
	int block_size = 32;
	
	int a_height = filter.N;
	int a_width = filter.C*filter.D*filter.H*filter.W;
	int b_height = input.row_num;
	int b_width = input.col_num;

	int* non_zero_vectors_dev;
	hipMalloc((void**)&non_zero_vectors_dev, sizeof(int)*number_of_non_zero_vectors);
	hipMemcpy(non_zero_vectors_dev, non_zero_vectors, sizeof(int)*number_of_non_zero_vectors, hipMemcpyHostToDevice);
	call_when_model_loaded(filter);

	output.row_num = a_height;
	output.col_num = b_width;
	output.nnz = a_height*number_of_non_zero_vectors;

	hipMalloc((void**)&output.ptr_dev, sizeof(int)*(output.col_num+1));
	hipMalloc((void**)&output.idx_dev, sizeof(int)*output.nnz);
	hipMalloc((void**)&output.val_dev, sizeof(float)*output.nnz);
	hipMemset(output.ptr_dev, 0, sizeof(int)*(output.col_num+1));
	printf("%d %d\n",a_height, a_width);
	printf("%d %d\n",b_height, b_width);
	printf("%d %d %d\n",output.row_num, output.col_num, output.nnz);
	printf("bs %d bn %d\n",block_size, block_num);	
	
	dense_sparse_mm<<<block_num, block_size >>>(a_height, a_width,filter.data_dev,
		input.ptr_dev, input.idx_dev, input.val_dev,
		output.ptr_dev,  output.idx_dev, output.val_dev,
		non_zero_vectors_dev);
	
	output.ptr = new int[output.col_num+1];
	output.idx = new int[output.nnz];
	output.val = new float[output.nnz];
	
	hipMemcpy(output.ptr, output.ptr_dev, sizeof(int)*(output.col_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(output.idx, output.idx_dev, sizeof(int)*output.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(output.val, output.val_dev, sizeof(float)*output.nnz,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	Mat temp;
	temp.row_num = output.row_num;
	temp.col_num = output.col_num;
	temp.coo = new COO[output.nnz];
	temp.nnz = output.nnz;
	int temp_idx = 0;
	for(int i=0; i<output.col_num+1; i++)
	{
		int nnz_per_col = output.ptr[i+1] ;
		int offset = output.ptr[i];
		for(int j=0;j<nnz_per_col;j++)
		{
			temp.coo[temp_idx].row = output.idx[offset+j];
			temp.coo[temp_idx].col = i;
			temp.coo[temp_idx].val = output.val[offset+j];
			temp_idx++;
		}
	}
	qsort(temp.coo, temp.nnz, sizeof(COO),compare2);
	print_coo(temp);
	
}

void sparse_conv_cuda(Mat& input, Mat& filter, Param& p, Mat& output)
{
	cube_to_coo_cuda(input, filter, p);
	int* non_zero_vectors = new int[input.col_num+1];
	int number_of_non_zero_vectors = 0;
	coo_to_csc_cuda(input, number_of_non_zero_vectors, non_zero_vectors);
	printf("number of non zero vectors : %d\n",number_of_non_zero_vectors);
	dense_sparse_mm_cuda(input, filter, output, number_of_non_zero_vectors, non_zero_vectors);

	
}
