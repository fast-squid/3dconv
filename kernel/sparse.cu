#include "hip/hip_runtime.h"
#include "mat.h"
#include <iostream>
#include <stdlib.h>
#define ERROR_CHECK \
{\
	hipError_t err = hipGetLastError(); \
	if ( hipSuccess != err ) \
	{\
		printf("[%s:%d]CUDA ERROR : %s\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
		exit(-1); \
	}\
}
__device__ __constant__ float eps = 0.0001;

// implementation without convolution parameter(padding, stride, groups, dilation)
__global__ void cube_to_coo(int input_d, int input_h, int input_w, float* input_data,
		int output_d, int output_h, int output_w, COO* coo, int *nnz, int stride)
{
    // block[8] : 0 ~ 7
	//printf(".\n");	
	if(threadIdx.x <5 && threadIdx.y < 5 && threadIdx.z <5)
	{
		
		int input_w_idx = (blockIdx.x % output_w)*stride + threadIdx.x;
		int input_h_idx = ((blockIdx.x / output_w) % output_h)*stride + threadIdx.y;
		int input_d_idx = (((blockIdx.x / output_w) / output_d) % output_d)*stride + threadIdx.z;

		int input_idx =  input_d_idx*(input_h*input_w)
			+ input_h_idx*(input_w)
			+ input_w_idx;
		if(input_data[input_idx]>eps)
		{
			int global_idx = atomicAdd(nnz, 1);
			coo[global_idx].row = threadIdx.z*(25)
				+ threadIdx.y*5
				+ threadIdx.x;
			coo[global_idx].col = blockIdx.x;
			coo[global_idx].val = input_data[input_idx];
		}
	}
	
}

int compare(const void* a, const void* b)
{
	COO x = *(COO*)a;
	COO y = *(COO*)b;
	if(x.col == y.col) return x.row - y.row;
	else return x.col - y.col;
}

void cube_to_coo_cuda(Mat& input, Mat& filter, Param& p)
{
	int output_N = 1;
	int output_C = filter.N;
	int output_D = 1+(input.D - filter.D + 2*p.padding)/p.stride;
	int output_H = 1+(input.H - filter.H + 2*p.padding)/p.stride;
	int output_W = 1+(input.W - filter.W + 2*p.padding)/p.stride;
	
	input.row_num = filter.C*filter.D*filter.H*filter.W;
	input.col_num = output_D*output_H*output_W;

	printf("input shape  : (%d,%d,%d,%d,%d)\n",input.N, input.C, input.D, input.H, input.W);
	printf("output shape : (%d,%d,%d,%d,%d)\n",output_N, output_C, output_D, output_H, output_W);
	printf("im2col shape : (%d,%d)\n",input.row_num, input.col_num);
	// coo format
	int* nnz_d;

	hipMalloc((void**)&input.coo_dev,sizeof(COO)*400000);
	hipMalloc((void**)&nnz_d, sizeof(int));
	hipMemset(nnz_d, 0, sizeof(int));
	
	// 3D-input
	int input_size = input.N*input.C*input.D*input.H*input.W;
	hipMalloc((void**)&input.data_dev, sizeof(float)*input_size);
	hipMemcpy(input.data_dev, input.data, sizeof(float)*input_size,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	int block_num = output_D*output_H*output_W;
	dim3 block_size(8,8,8);
	
	cube_to_coo<<<block_num, block_size>>>
		(input.D, input.H, input.W, input.data_dev,
		 output_D, output_H, output_W,input.coo_dev,
		 nnz_d,p.stride);
	ERROR_CHECK;	

	hipMemcpy(&input.nnz,nnz_d, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	input.coo = new COO[input.nnz];
	hipMemcpy(input.coo, input.coo_dev, input.nnz*sizeof(COO),  hipMemcpyDeviceToHost);
	qsort(input.coo, input.nnz, sizeof(COO),compare);
	hipMemcpy(input.coo_dev, input.coo, input.nnz*sizeof(COO),  hipMemcpyHostToDevice);

	// cudafree
	hipFree(input.data_dev);
}

__global__ void coo_2_csr(COO* coo, 
		int* ptr, int *idx, float* val,
		int nnz)
{
    int global_tid = threadIdx.x+blockDim.x*blockIdx.x;
	if(global_tid<nnz)
	{
		idx[global_tid] = coo[global_tid].col;
		val[global_tid] = coo[global_tid].val;
		atomicAdd(&ptr[coo[global_tid].row+1], 1);
	}
}

void coo_to_csr_cuda(Mat& input)
{
	hipMalloc((void**)&input.ptr_dev, sizeof(int)*(input.row_num+1));
	hipMalloc((void**)&input.idx_dev, sizeof(int)*input.nnz);
	hipMalloc((void**)&input.val_dev, sizeof(float)*input.nnz);
	hipMemset(input.ptr_dev, 0, sizeof(int)*(input.row_num+1));

	int block_num = input.nnz/1024+1;
	int block_size = 1024;
	
	coo_2_csr<<<block_num, block_size>>>(input.coo_dev,
			input.ptr_dev, input.idx_dev, input.val_dev,
			input.nnz);
	ERROR_CHECK;	
	
	input.ptr = new int[input.row_num+1];
	input.idx = new int[input.nnz];
	input.val = new float[input.nnz];

	hipMemcpy(input.ptr, input.ptr_dev, sizeof(int)*(input.row_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(input.idx, input.idx_dev, sizeof(int)*input.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(input.val, input.val_dev, sizeof(float)*input.nnz,hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	for(int i=0;i<input.row_num;i++)
	{
		input.ptr[i+1] += input.ptr[i];
	}
	printf("nnz %d %d\n",input.nnz, input.ptr[input.row_num]);
	hipMemcpy(input.ptr_dev, input.ptr, sizeof(int)*(input.row_num+1),hipMemcpyHostToDevice);
}

__global__ void coo_2_csc(COO* coo, 
		int* ptr, int *idx, float* val,
		int nnz)
{
    int global_tid = threadIdx.x+blockDim.x*blockIdx.x;
	if(global_tid<nnz)
	{
		idx[global_tid] = coo[global_tid].row;
		val[global_tid] = coo[global_tid].val;
		atomicAdd(&ptr[coo[global_tid].col+1], 1);
	}
}

void coo_to_csc_cuda(Mat& input, int& number_of_non_zero_vectors, int *non_zero_vectors)
{
	hipMalloc((void**)&input.ptr_dev, sizeof(int)*(input.col_num+1));
	hipMalloc((void**)&input.idx_dev, sizeof(int)*input.nnz);
	hipMalloc((void**)&input.val_dev, sizeof(float)*input.nnz);
	hipMemset(input.ptr_dev, 0, sizeof(int)*(input.col_num+1));

	int block_num = input.nnz/1024+1;
	int block_size = 1024;
	
	coo_2_csc<<<block_num, block_size>>>(input.coo_dev,
			input.ptr_dev, input.idx_dev, input.val_dev,
			input.nnz);
	ERROR_CHECK;	
	
	input.ptr = new int[input.col_num+1];
	input.idx = new int[input.nnz];
	input.val = new float[input.nnz];

	hipMemcpy(input.ptr, input.ptr_dev, sizeof(int)*(input.col_num+1),hipMemcpyDeviceToHost);
	hipMemcpy(input.idx, input.idx_dev, sizeof(int)*input.nnz,hipMemcpyDeviceToHost);
	hipMemcpy(input.val, input.val_dev, sizeof(float)*input.nnz,hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	for(int i=0;i<input.col_num;i++)
	{
		input.ptr[i+1] += input.ptr[i];
		if(input.ptr[i+1]-input.ptr[i])
		{
			non_zero_vectors[number_of_non_zero_vectors] = i;
			number_of_non_zero_vectors++;
		}
	}

	printf("nnz %d %d\n",input.nnz, input.ptr[input.col_num]);
	hipMemcpy(input.ptr_dev, input.ptr, sizeof(int)*(input.col_num+1),hipMemcpyHostToDevice);
}



//__global__ void dense_sparse_mm(int a_height, int a_width, float* a_val,
//		int* b_ptr, int* b_idx, float* b_val,
//		int* c_ptr, int* c_idx, float* c_val,
//		int* non_zero_bin)
//{
//	int col_idx = aaaaaaa[blockIdx.x];
//	int row_offset = b_ptr[col_idx];
//	
//	// load B's row idx to shared memory
//	__shared__ int smem_b_row[1024];
//	__shared__ int smem_b_val[1024];
//
//	for(int tid = threadIdx.x; tid < nnz; tid+=blockDim.x)
//	{
//		smem_b_row[tid] = b_idx[row_offset + tid];
//		smem_b_val[tid] = b_val[row_offset + tid];
//	}
//	__syncthreads();
//
//	__shared__ float smem_c_idx[];
//	__shared__ float smem_c_val[1024];
//	for(int a_h = threadIdx.x; a_h < a_height; a_h+=blockDim.x)
//	{
//		int idx = s_mem_b_row[tid];
//		c_idx[a_h] = a_h;
//		for(int idx = 0; idx < nnz; idx++)
//		{
//			smem_c_val[a_h] += a_val[a_h*a_width + idx] * smem_b_val[idx];
//		}
//	}
//	c_ptr[col_idx+1] = a_height*blockIdx.x;
//}
//
//void dense_sparse_mm_cuda(Mat& input, Mat& filter, Mat& output,
//		int number_of_non_zero_vectors, const int& non_zero_vectors)
//{
//	int block_size;
//	int block_num;
//	
//	int a_height = filter.N;
//	int a_width = filter.C*filter.D*filter.H*filter.W;
//	
//	dense_sparse_mm<<<block_num, block_size >>>(a_height, a_width ,filter.data_dev,
//		input.ptr_dev, input.idx_dev, input.val_dev,
//		output.ptr_dev,  output.idx_dev, output.val_ev,
//		int* non_zero_bin);
//}
//*/
void sparse_conv_cuda(Mat& input, Mat& filter, Param& p, Mat& output)
{
	cube_to_coo_cuda(input, filter, p);
	int* non_zero_vectors = new int[input.col_num+1];
	int number_of_non_zero_vectors = 0;
	coo_to_csc_cuda(input, number_of_non_zero_vectors, non_zero_vectors);
	printf("number of non zero vectors : %d\n",number_of_non_zero_vectors);
	//dense_sparse_mm_cuda(input, filter, output, );

	
}
