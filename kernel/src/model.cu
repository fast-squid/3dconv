#include "model.h"

#define ERROR_CHECK \
{\
	hipError_t err = hipGetLastError(); \
	if ( hipSuccess != err ) \
	{\
		printf("[%s:%d]CUDA ERROR : %s\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
	}\
}

void Model::set_cuda(bool transpose)
{	
	Mat* filter_ptr = 0;
	for(int i=0;i < inner_layers.size();i++)
	{
		if(inner_layers[i]->type == CONV)
		{
			filter_ptr = &((Conv*)inner_layers[i])->filter;
			int size = filter_ptr->get_mat_size();
			float* data_host;
			if(transpose)
			{
				filter_ptr->set_transpose();
				data_host = filter_ptr->data_trans;
			}
			else
			{
				data_host = filter_ptr->data;
			}
			hipMalloc((void**)&filter_ptr->data_dev, sizeof(float)*size);
			hipMemcpy(filter_ptr->data_dev, data_host, sizeof(float)*size, hipMemcpyHostToDevice); 
		}
	}
}

